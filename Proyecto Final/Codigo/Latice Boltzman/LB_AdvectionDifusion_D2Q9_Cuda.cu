#include "hip/hip_runtime.h"
//LB for Waves D2Q5 on CUDA
#include <iostream>
#include <fstream>
#include <cmath>
using namespace std;

#define Lx 128
#define Ly 128
#define N 32 //Threads per Block
const int M=(Lx*Ly+N-1)/N; //Blocks per Grid
#define Q 5
const int ArraySize=Lx*Ly*Q;

const float W0=1.0/3;

const float C=0.5; // C<0.707 cells/click
const float C2=C*C;
const float AUX0=1-3*C2*(1-W0);

const float tau=0.5;
const float Utau=1.0/tau;
const float UmUtau=1-Utau;

//------------ PROGRAMMING ON THE DEVICE ----------------
//---------------Constants (Symbols)----------------
__constant__ float d_w[5];
__constant__ int d_Vx[5];
__constant__ int d_Vy[5];
__constant__ float d_C[3];   // d_C[0]=C,  d_C[1]=C2,  d_C[2]=AUX, 
__constant__ float d_tau[3]; // d_tau[0]=tau,  d_tau[1]=Utau,  d_tau[2]=UmUtau, 
//----------Functions called by the device itself
//Data index
__device__ int d_n(int ix,int iy,int i){
  return ix*Ly*Q+iy*Q+i;  
}
//Macroscopic Fields
__device__ float d_rho(int ix,int iy,float *d_f){
  float sum=0; int i,n0;
  for(i=0;i<Q;i++){
    n0=d_n(ix,iy,i); sum+=d_f[n0];
  }
  return sum;
}
__device__ float d_Jx(int ix,int iy,float *d_f){
  float sum=0; int i,n0;
  for(i=0;i<Q;i++){
    n0=d_n(ix,iy,i); sum+=d_Vx[i]*d_f[n0];
  }
  return sum;
}
__device__ float d_Jy(int ix,int iy,float *d_f){
  float sum=0; int i,n0;
  for(i=0;i<Q;i++){
    n0=d_n(ix,iy,i); sum+=d_Vy[i]*d_f[n0];
  }
  return sum;
}
//Equilibrium Functions
__device__ float d_feq(float rho0,float Jx0,float Jy0,int i){
  return 3*d_w[i]*(d_C[1]*rho0+d_Vx[i]*Jx0+d_Vy[i]*Jy0);
}
__device__ float d_f0eq(float rho0,float Jx0,float Jy0){
  return rho0*d_C[2];
}
//---------------------KERNELS----------------------------
__global__ void d_Collision(float *d_f,float *d_fnew,float *d_test){
  //Define internal registers
  int icell,ix,iy,i,n0;  float rho0,Jx0,Jy0;
  //Find which thread an which cell should I work
  icell=blockIdx.x*blockDim.x+threadIdx.x;
  ix=icell/Ly; iy=icell%Ly;
  //Compute the macroscopic fields
  rho0=d_rho(ix,iy,d_f); //rho
  Jx0=d_Jx(ix,iy,d_f);   //Jx0
  Jy0=d_Jy(ix,iy,d_f);   //Jy0
  if (ix==Lx/4 && iy== Lx/3) d_test[0]=Jy0; //OJO
  //Collide and compute fnew
  n0=d_n(ix,iy,0); d_fnew[n0]=d_tau[2]*d_f[n0]+d_tau[1]*d_f0eq(rho0,Jx0,Jy0);
  for(i=1;i<Q;i++){ //on each direction
    n0=d_n(ix,iy,i); d_fnew[n0]=d_tau[2]*d_f[n0]+d_tau[1]*d_feq(rho0,Jx0,Jy0,i);
  }
}
__global__ void d_ImposeFields(float *d_f,float *d_fnew,float RhoSource){
  //Define internal registers
  int ix,iy,i,n0;  float rho0,Jx0,Jy0;
  //There is only one thread and for one cell in the center
  ix=Lx/2; iy=Ly/2;
  //Compute the macroscopic fields
  rho0=RhoSource; //rho
  Jx0=d_Jx(ix,iy,d_f);   //Jx0
  Jy0=d_Jy(ix,iy,d_f);   //Jy0
  //Collide and compute fnew
  n0=d_n(ix,iy,0); d_fnew[n0]=d_f0eq(rho0,Jx0,Jy0);
  for(i=1;i<Q;i++){ //on each direction
    n0=d_n(ix,iy,i); d_fnew[n0]=d_feq(rho0,Jx0,Jy0,i);
  }
}
__global__ void d_Advection(float *d_f,float *d_fnew){
  //Define internal registers
  int icell,ix,iy,i,ixnext,iynext,n0,n0next;
  //Find which thread an which cell should I work
  icell=blockIdx.x*blockDim.x+threadIdx.x;
  ix=icell/Ly; iy=icell%Ly;
  //Move the contents to the neighboring cells
  for(i=0;i<Q;i++){ //on each direction
    ixnext=(ix+d_Vx[i]+Lx)%Lx; iynext=(iy+d_Vy[i]+Ly)%Ly;//periodic boundaries
    n0=d_n(ix,iy,i); n0next=d_n(ixnext,iynext,i);
    d_f[n0next]=d_fnew[n0]; 
  }
}
//------------ PROGRAMMING ON THE HOST ----------------
//-------------LatticeBoltzmann class------------
class LatticeBoltzmann{
private:
  float h_C[3];   // h_C[0]=C,  h_C[1]=C2,  h_C[2]=AUX, 
  float h_tau[3]; // h_tau[0]=tau,  h_tau[1]=Utau,  h_tau[2]=UmUtau, 
  float h_w[5]; // w[i]
  int h_Vx[5],h_Vy[5]; // Vx[i],Vy[i]
  float *h_f,*h_fnew;  float *d_f,*d_fnew;// f[ix][iy][i]
  float *h_Test,*d_Test; //Just for tests
public:
  LatticeBoltzmann(void);
  ~LatticeBoltzmann(void);
  int n(int ix,int iy,int i){return (ix*Ly+iy)*Q+i;};
  float h_rho(int ix,int iy);
  float h_feq(float rho0,float Jx0,float Jy0,int i);
  void Start(float rho0,float Jx0,float Jy0);
  void Collision(void);
  void ImposeFields(int t);
  void Advection(void);
  void Print(const char * NameFile);
  void ShowTest(void);
};  
LatticeBoltzmann::LatticeBoltzmann(void){
  //CONSTANTS(d_Symbols)
  //---Charge constantes on the Host-----------------
  //running constants
  h_C[0]=C;  h_C[1]=C2;  h_C[2]=AUX0;
  h_tau[0]=tau;  h_tau[1]=Utau;  h_tau[2]=UmUtau;
  //weights
  h_w[0]=W0; h_w[1]=h_w[2]=h_w[3]=h_w[4]=(1.0-W0)/4;
  //velocity vectors
  h_Vx[0]=0;  h_Vx[1]=1;  h_Vx[2]=0;  h_Vx[3]=-1; h_Vx[4]=0;  
  h_Vy[0]=0;  h_Vy[1]=0;  h_Vy[2]=1;  h_Vy[3]=0;  h_Vy[4]=-1;
  //------Send to the Device-----------------
  hipMemcpyToSymbol(HIP_SYMBOL(d_w),h_w,Q*sizeof(float),0,hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(d_Vx),h_Vx,Q*sizeof(int),0,hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(d_Vy),h_Vy,Q*sizeof(int),0,hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(d_C),h_C,3*sizeof(float),0,hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(d_tau),h_tau,3*sizeof(float),0,hipMemcpyHostToDevice);
  //DISTRIBUTION FUNCTIONS
  //Build the dynamic matrices on the host
  h_f=new float [ArraySize];  h_fnew=new float [ArraySize];
  //Build the dynamic matrices on the device
  hipMalloc((void**) &d_f,ArraySize*sizeof(float));
  hipMalloc((void**) &d_fnew,ArraySize*sizeof(float));
  //Test variables
   h_Test=new float [1]; hipMalloc((void**) &d_Test,sizeof(float));
}
LatticeBoltzmann::~LatticeBoltzmann(void){
  delete[] h_f;  delete[] h_fnew;
  hipFree(d_f);  hipFree(d_fnew);
  //Test variables
  delete[] h_Test; hipFree(d_Test);
}
float LatticeBoltzmann::h_rho(int ix,int iy){
  //Note: Please import data from device before running
  float sum; int i,n0;
  for(sum=0,i=0;i<Q;i++){
    n0=n(ix,iy,i); sum+=h_fnew[n0];
  }
  return sum;
}
float LatticeBoltzmann::h_feq(float rho0,float Jx0,float Jy0,int i){
  if(i>0)
    return 3*h_w[i]*(C2*rho0+h_Vx[i]*Jx0+h_Vy[i]*Jy0);
  else
    return rho0*AUX0;
}
void LatticeBoltzmann::Start(float rho0,float Jx0,float Jy0){
  int ix,iy,i,n0;
  //Charge on the Host
  for(ix=0;ix<Lx;ix++) //for each cell
    for(iy=0;iy<Ly;iy++)
      for(i=0;i<Q;i++){ //on each direction
	n0=n(ix,iy,i); h_f[n0]=h_feq(rho0,Jx0,Jy0,i);
      }
  //Send to the Device
  hipMemcpy(d_f,h_f,ArraySize*sizeof(float),hipMemcpyHostToDevice);
}  
void LatticeBoltzmann::Collision(void){
  //Do everything on the Device
  dim3 ThreadsPerBlock(N,1,1);
  dim3 BlocksPerGrid(M,1,1);
  d_Collision<<<BlocksPerGrid,ThreadsPerBlock>>>(d_f,d_fnew,d_test); //OJO, quitar test
  hipMemcpy(h_Test,d_Test,sizeof(float),hipMemcpyDeviceToHost); //OJO
  cout<<"Test="<<h_Test[0]<<endl; //OJO
}
void LatticeBoltzmann::ImposeFields(int t){
  float lambda=10, omega=2*M_PI/lambda*C;
  float RhoSource=10*sin(omega*t);
  dim3 ThreadsPerBlock(1,1,1); //A single thread (in this case)
  dim3 BlocksPerGrid(1,1,1);
  d_ImposeFields<<<BlocksPerGrid,ThreadsPerBlock>>>(d_f,d_fnew,RhoSource);
}
void LatticeBoltzmann::Advection(void){
  //Do everything on the Device
  dim3 ThreadsPerBlock(N,1,1);
  dim3 BlocksPerGrid(M,1,1);
  d_Advection<<<BlocksPerGrid,ThreadsPerBlock>>>(d_f,d_fnew);
}
void LatticeBoltzmann::Print(const char * NameFile){
  ofstream MyFile(NameFile); double rho0; int ix,iy;
  //Bring back the data from Device to Host
  hipMemcpy(h_fnew,d_fnew,ArraySize*sizeof(float),hipMemcpyDeviceToHost);
  //Print for gnuplot splot
  for(ix=0;ix<Lx;ix++){
    for(iy=0;iy<Ly;iy++){
      rho0=h_rho(ix,iy);
      MyFile<<ix<<" "<<iy<<" "<<rho0<<endl;
    }
    MyFile<<endl;
  }
  MyFile.close();
}
void LatticeBoltzmann::ShowTest(void){
  //Bring back test data from Device to Host
  hipMemcpy(h_Test,d_Test,sizeof(float),hipMemcpyDeviceToHost);
  cout<<"Test="<<h_Test[0]<<endl;
}

//--------------- GLOBAL FUNCTIONS ------------

int main(void){
  LatticeBoltzmann Waves;
  int t,tmax=100;
  float rho0=0,Jx0=0,Jy0=0;

  //Start
  Waves.Start(rho0,Jx0,Jy0);
  //Run
  for(t=0;t<tmax;t++){
    Waves.Collision();
    Waves.ImposeFields(t);
    Waves.Advection();
  }
  //Print Results
  Waves.Print("Waves_CUDA.dat");
 
  return 0;
}  
