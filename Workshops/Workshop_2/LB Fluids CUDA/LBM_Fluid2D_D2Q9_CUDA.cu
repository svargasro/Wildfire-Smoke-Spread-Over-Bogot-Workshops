// LB for Fluids D2Q9 on CUDA

#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cmath>
using namespace std;

#define Lx 256
#define Ly 64
#define N 32 // Threads per Block
const int M=(Lx*Ly+N-1)/N; // Blocks per Grid
#define Q 9
const int ArraySize=Lx*Ly*Q;

const float tau=0.55;
const float Utau=1.0/tau;
const float UmUtau=1-Utau;

//------------ PROGRAMMING ON THE DEVICE ----------------
//---------------Constants (Symbols)----------------
__constant__ float d_w[9];
__constant__ int d_Vx[9];
__constant__ int d_Vy[9];
__constant__ float d_tau[3]; // d_tau[0]=tau,  d_tau[1]=Utau,  d_tau[2]=UmUtau,

//----------Functions called by the device itself
__device__ int d_n(int ix, int iy, int i) {
    return ix * Ly * Q + iy * Q + i;
}

__device__ float d_rho(int ix, int iy, float *d_f) {
    float sum = 0; int i, n0;
    for (i = 0; i < Q; i++) {
        n0 = d_n(ix, iy, i); sum += d_f[n0];
    }
    return sum;
}

__device__ float d_Jx(int ix, int iy, float *d_f) {
    float sum = 0; int i, n0;
    for (i = 0; i < Q; i++) {
        n0 = d_n(ix, iy, i); sum += d_Vx[i] * d_f[n0];
    }
    return sum;
}

__device__ float d_Jy(int ix, int iy, float *d_f) {
    float sum = 0; int i, n0;
    for (i = 0; i < Q; i++) {
        n0 = d_n(ix, iy, i); sum += d_Vy[i] * d_f[n0];
    }
    return sum;
}

__device__ float d_feq(float rho0, float Ux0, float Uy0, int i) {
    float UdotVi = Ux0 * d_Vx[i] + Uy0 * d_Vy[i], U2 = Ux0 * Ux0 + Uy0 * Uy0;
    return rho0 * d_w[i] * (1 + 3 * UdotVi + 4.5 * UdotVi * UdotVi - 1.5 * U2);
}

//---------------------KERNELS----------------------------
__global__ void d_Collision(float *d_f, float *d_fnew) {
    int icell, ix, iy, i, n0; float rho0, Ux0, Uy0;
    icell = blockIdx.x * blockDim.x + threadIdx.x;
    ix = icell / Ly; iy = icell % Ly;
    if (icell < Lx * Ly) {
        rho0 = d_rho(ix, iy, d_f);
        Ux0 = d_Jx(ix, iy, d_f) / rho0;
        Uy0 = d_Jy(ix, iy, d_f) / rho0;
        for (i = 0; i < Q; i++) {
            n0 = d_n(ix, iy, i);
            d_fnew[n0] = d_tau[2] * d_f[n0] + d_tau[1] * d_feq(rho0, Ux0, Uy0, i);
        }
    }
}

__global__ void d_ImposeFields(float *d_f, float *d_fnew, float Ufan) {
    int ix, iy, i, n0, ixc = Lx / 8, iyc = Ly / 2, R = Ly / 5; float rho0;
    int icell = blockIdx.x * blockDim.x + threadIdx.x;
    ix = icell / Ly; iy = icell % Ly;
    float R2 = R * R;
    if (icell < Lx * Ly) {
        rho0 = d_rho(ix, iy, d_f);
        if (ix == 0) {
            for (i = 0; i < Q; i++) { n0 = d_n(ix, iy, i); d_fnew[n0] = d_feq(rho0, Ufan, 0, i); }
        } else if ((ix - ixc) * (ix - ixc) + (iy - iyc) * (iy - iyc) <= R2) {
            for (i = 0; i < Q; i++) { n0 = d_n(ix, iy, i); d_fnew[n0] = d_feq(rho0, 0, 0, i); }
        } else if (ix == ixc && iy == iyc + R + 1) {
            for (i = 0; i < Q; i++) { n0 = d_n(ix, iy, i); d_fnew[n0] = d_feq(rho0, 0, 0, i); }
        }
    }
}

__global__ void d_Advection(float *d_f, float *d_fnew) {
    int icell, ix, iy, i, ixnext, iynext, n0, n0next;
    icell = blockIdx.x * blockDim.x + threadIdx.x;
    ix = icell / Ly; iy = icell % Ly;
    if (icell < Lx * Ly) {
        for (i = 0; i < Q; i++) {
            ixnext = (ix + d_Vx[i] + Lx) % Lx;
            iynext = (iy + d_Vy[i] + Ly) % Ly;
            n0 = d_n(ix, iy, i); n0next = d_n(ixnext, iynext, i);
            d_f[n0next] = d_fnew[n0];
        }
    }
}

//------------ PROGRAMMING ON THE HOST ----------------
//-------------LatticeBoltzmann class------------

class LatticeBoltzmann {
private:
    float h_tau[3];
    float h_w[9];
    int h_Vx[9], h_Vy[9];
    float *h_f, *h_fnew;  float *d_f, *d_fnew;
public:
    LatticeBoltzmann(void);
    ~LatticeBoltzmann(void);
    int h_n(int ix, int iy, int i) { return (ix * Ly + iy) * Q + i; }
    float h_rho(int ix, int iy);
    float h_Jx(int ix, int iy);
    float h_Jy(int ix, int iy);
    float h_feq(float rho0, float Ux0, float Uy0, int i);
    void Start(float rho0, float Ux0, float Uy0);
    void Collision(void);
    void ImposeFields(float Ufan);
    void Advection(void);
    void Print(const char * NameFile, float Ufan);
};

LatticeBoltzmann::LatticeBoltzmann(void) {

    // Inicialización de las constantes en el host
    h_tau[0] = tau;  h_tau[1] = Utau;  h_tau[2] = UmUtau;
    h_w[0] = 4.0 / 9; h_w[1] = h_w[2] = h_w[3] = h_w[4] = 1.0 / 9;
    h_w[5] = h_w[6] = h_w[7] = h_w[8] = 1.0 / 36;
    h_Vx[0] = 0;  h_Vx[1] = 1;  h_Vx[2] = 0;  h_Vx[3] = -1; h_Vx[4] = 0;
    h_Vx[5] = 1;  h_Vx[6] = -1; h_Vx[7] = -1; h_Vx[8] = 1;
    h_Vy[0] = 0;  h_Vy[1] = 0;  h_Vy[2] = 1;  h_Vy[3] = 0;  h_Vy[4] = -1;
    h_Vy[5] = 1;  h_Vy[6] = 1;  h_Vy[7] = -1; h_Vy[8] = -1;

    // Copia de constantes a la memoria de la GPU
    hipMemcpyToSymbol(HIP_SYMBOL(d_w), h_w, Q * sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_Vx), h_Vx, Q * sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_Vy), h_Vy, Q * sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_tau), h_tau, 3 * sizeof(float), 0, hipMemcpyHostToDevice);

// Asignación de memoria para las funciones de distribución en el host
    h_f = new float[ArraySize];  h_fnew = new float[ArraySize];

    // Asignación de memoria para las funciones de distribución en la GPU
    hipMalloc((void**)&d_f, ArraySize * sizeof(float));
    hipMalloc((void**)&d_fnew, ArraySize * sizeof(float));
}
//---------------------Destructor---------------------
LatticeBoltzmann::~LatticeBoltzmann(void){
    delete[] h_f;  delete[] h_fnew;
    hipFree(d_f);  hipFree(d_fnew);
}

//---------------------Start---------------------
// Inicializa las funciones de distribución en el host
void LatticeBoltzmann::Start(float rho0, float Ux0, float Uy0) {
    int ix, iy, i, n0;
    for (ix = 0; ix < Lx; ix++) {
        for (iy = 0; iy < Ly; iy++) {
            for (i = 0; i < Q; i++) {
                n0 = h_n(ix, iy, i);
                h_f[n0] = h_feq(rho0, Ux0, Uy0, i);
            }
        }
    }

    // Copia las funciones de distribución inicializadas a la GPU
    hipMemcpy(d_f, h_f, ArraySize * sizeof(float), hipMemcpyHostToDevice);
}
//---------------------Campos macroscópicos---------------------
float LatticeBoltzmann::h_rho(int ix,int iy){
  //Note: Please import data from device before running
  float sum; int i,n0;
  for(sum=0,i=0;i<Q;i++){
    n0=h_n(ix,iy,i); sum+=h_fnew[n0];
  }
  return sum;
}
float LatticeBoltzmann::h_Jx(int ix,int iy){
  //Note: Please import data from device before running
  float sum; int i,n0;
  for(sum=0,i=0;i<Q;i++){
    n0=h_n(ix,iy,i); sum+=h_Vx[i]*h_fnew[n0];
  }
  return sum;
}
float LatticeBoltzmann::h_Jy(int ix,int iy){
  //Note: Please import data from device before running
  float sum; int i,n0;
  for(sum=0,i=0;i<Q;i++){
    n0=h_n(ix,iy,i); sum+=h_Vy[i]*h_fnew[n0];
  }
  return sum;
}

//---------------------Función de equilibrio---------------------
float LatticeBoltzmann::h_feq(float rho0, float Ux0, float Uy0, int i) {
    float UdotVi = Ux0 * h_Vx[i] + Uy0 * h_Vy[i], U2 = Ux0 * Ux0 + Uy0 * Uy0;
    return rho0 * h_w[i] * (1 + 3 * UdotVi + 4.5 * UdotVi * UdotVi - 1.5 * U2);
}

//---------------------Reglas de evolución---------------------

void LatticeBoltzmann::Collision(void) {
    dim3 ThreadsPerBlock(N, 1, 1);
    dim3 BlocksPerGrid(M, 1, 1);
    d_Collision<<<BlocksPerGrid, ThreadsPerBlock>>>(d_f, d_fnew);
}

void LatticeBoltzmann::ImposeFields(float Ufan) {
    dim3 ThreadsPerBlock(N, 1, 1);
    dim3 BlocksPerGrid(M, 1, 1);
    d_ImposeFields<<<BlocksPerGrid, ThreadsPerBlock>>>(d_f, d_fnew, Ufan);
}

void LatticeBoltzmann::Advection(void) {
    dim3 ThreadsPerBlock(N, 1, 1);
    dim3 BlocksPerGrid(M, 1, 1);
    d_Advection<<<BlocksPerGrid, ThreadsPerBlock>>>(d_f, d_fnew);
}

void LatticeBoltzmann::Print(const char * NameFile, float Ufan) {

    ofstream MyFile(NameFile);
    double rho0, Ux0, Uy0;
    int ix, iy;

    hipMemcpy(h_fnew, d_fnew, ArraySize * sizeof(float), hipMemcpyDeviceToHost);

    for (ix = 0; ix < Lx; ix += 4) {
        for (iy = 0; iy < Ly; iy += 4) {
            //rho0 = h_rho(ix, iy); Ux0 = h_Jx(ix, iy, true) / rho0; Uy0 = h_Jy(ix, iy, true) / rho0;
            rho0 = h_rho(ix, iy); Ux0 = h_Jx(ix, iy) / rho0; Uy0 = h_Jy(ix, iy) / rho0;
            MyFile << ix << " " << iy << " " << Ux0 / Ufan * 4 << " " << Uy0 / Ufan * 4 << endl;
        }
        MyFile << endl;
    }
    MyFile.close();
}

//--------------- GLOBAL FUNCTIONS ------------
int main(void) {
    LatticeBoltzmann Air;
    int t, tmax = 10000;
    float rho0 = 1.0, Ufan0 = 0.1;

    Air.Start(rho0, Ufan0, 0);
    for (t = 0; t < tmax; t++) {
        Air.Collision();
        Air.ImposeFields(Ufan0);
        Air.Advection();
    }
    //Air.Print("WindChannel.dat", Ufan0);

    return 0;
}
